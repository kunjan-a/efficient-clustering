#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil.h>
#include <settings_xy.h>
//#include <cuda_kernel_xy.cu>
//#include <cuda_kernel_xy_tree.cu>
//#include <cuda_kernel_xy_multi.cu>
#include <cuda_kernel_xy_combined.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
__host__ void mean_by_CUDA( int n, int max_iter, int k, float threshold, float *h_dataptx, float *h_datapty);

////////////////////////////////////////////////////////////////////////////////
//! Global Memory size required in bytes rounded to nearest multiple of mem. segment size.
//! The rules followed by 1.2 and 1.3 CUDA cards are:
//! The following protocol is used to determine the memory transactions
//! necessary to service all threads in a half-warp:
//!
//! Find the memory segment that contains the address requested by the lowest
//! numbered active thread. The segment size depends on the size of the words
//! accessed by the threads:
//! 32 bytes for 1-byte words,
//! 64 bytes for 2-byte words,
//! 128 bytes for 4-, 8- and 16-byte words.
//!
//! Find all other active threads whose requested address lies in the same segment.
//! Reduce the transaction size, if possible:
//! If the transaction size is 128 bytes and only the lower or upper half is used,
//! reduce the transaction size to 64 bytes;
//! If the transaction size is 64 bytes (originally or after reduction from 128
//! bytes) and only the lower or upper half is used, reduce the transaction size
//! to 32 bytes.
//!
//! Carry out the transaction and mark the serviced threads as inactive.
//!
//! Repeat until all threads in the half-warp are serviced.
//! @param n size required
////////////////////////////////////////////////////////////////////////////////
template<class T>
__device__ __host__ unsigned int get_gmem_length(int n)
{
    int size=sizeof(T);
    int logSegLength=7;
    if(size==1)
    {
        logSegLength=5;
    }
    else
    {
        if(size==2)
            logSegLength=6;
    }
    // gives bytes rounded to nearest multiple of segment size i.e. ceil(mem_required/segmenSize)*segmentSize
    return ((((n*size-1)>>logSegLength) + 1) <<logSegLength);
}

////////////////////////////////////////////////////////////////////
//! We store the points in d arrays each of length (n + n%128), to ensure that all global memory accesses are aligned to transaction size.
//!
//! If points more than device memory then we can issue the kmeans code multiple times for each iteration and
//! finally add up the individual counts and sums from each such invocation to get new centroids. We have not coded for such case right now.
//!
//! Another way could be to store the points in following manner:
//! Store 1st co-ordinate of 'p' points, followed by their 2nd co-ordinate, followed by their 3rd co-ordinate and so on,
//! so that when first co-ordinate is read the 2nd co-ordinate comes into cache (assuming there is cache prefetching).
//! Here the number 'p' can be 'w', or 't', or t*b',
//! where b' is the number of blocks that can run simultaneously i.e. m*MIN_NUM_BLOCKS_PER_MULTIPROCESSOR.
///////////////////////////////////////////////////////////////////
__host__ unsigned int load_points_in_device(const unsigned int n, float* &d_dataptx, float* &d_datapty,float* &h_dataptx, float* &h_datapty)
{

    float *d_data;
    unsigned int memReq=get_gmem_length<float>(n);
    // TODO: Check if n is too large for global mempry to store at once.

    // Allcate device memory for data points and centroids. Store x-coord of all points followed by y-coord of all points
    // followed by x-coord of centroids followed by y-coord of centroids

    printf( "device memory required for storing %d points is: %d\n", n,(memReq<<1));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data, (memReq<<1)));

    unsigned int lenPerDim_d_datapt = memReq/sizeof(float);
    d_dataptx=d_data;                                                                          // all coord of datapoints
    d_datapty=d_dataptx + lenPerDim_d_datapt;
    printf("d_data:%u, d_dataptx:%u, d_datapty: %u \n",d_data,d_dataptx,d_datapty);

    CUDA_SAFE_CALL( hipMemcpy( d_dataptx, h_dataptx, n*sizeof(float), hipMemcpyHostToDevice) );           //copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_datapty, h_datapty, n*sizeof(float), hipMemcpyHostToDevice) );           //copy host memory to device

    return lenPerDim_d_datapt;
}


////////////////////////////////////////////////////////////////////
//! We select the first k points as the initial centroids.
//! We might as well take the points at k random positions but this decision would not effect
//! the difference b/w performance of openmp and CUDA as both will run for equal number of iterations for the same choice of k initial centroids.
//! In fact because of this instead of terminating kmeans when the cluster centres stop changing in subsequent iterations,
//! we let it run for a fixed number of iterations for both CUDA and openmp as the amount of work done in each iteration is same.
//!
//! We store the centroids in the device memory again in d arrays each of size (k + k%128)
//!
//! We could store them in constant memory too, provided it fits in its small size, to make use of constant cache.
//! But we invoke the clustering kernel only once and CUDA restricts changing contents of constant memory from inside a kernel.
//! Instead if we call the kernel separaely for each iteration then we can replace the old centroid values in the constant memory
//! with new centroids found at the end of the iteration.
///////////////////////////////////////////////////////////////////
__host__ unsigned int load_centroids_in_device(const unsigned int k, float * &d_centroidx, float * &d_centroidy, float * &d_dataptx, float * &d_datapty)
{

    float *d_data;
    unsigned int memReq=get_gmem_length<float>(k);

    printf( "device memory required for storing %d centroids is: %d\n", k,(memReq<<1));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data, (memReq<<1)));

    unsigned int lenPerDim_d_centroid=memReq/sizeof(float);
    d_centroidx=d_data;                                                                        // x-coord of centroids
    d_centroidy=d_data+lenPerDim_d_centroid;                                                            // y-coord of centroids
    printf("d_data:%u, d_centroidx: %u, d_centroidy:%u \n",d_data,d_centroidx,d_centroidy);

    CUDA_SAFE_CALL( hipMemcpy( d_centroidx, d_dataptx, sizeof(float)*k, hipMemcpyDeviceToDevice) );   //Initialize the first k data points as the k centroids.
    CUDA_SAFE_CALL( hipMemcpy( d_centroidy, d_datapty, sizeof(float)*k, hipMemcpyDeviceToDevice) );

    return lenPerDim_d_centroid;
}


////////////////////////////////////////////////////////////////////
//! The external reduction between blocks requires synchronisation among blocks.
//! We store an array of synchronisation vars which store the value of the iteration for which the block has finished reduction.
//! The initial value is -1 for all the blocks.
//! @return Base address of the synchronisation array which was created in the device memory
///////////////////////////////////////////////////////////////////
__host__ int* create_block_synchronisn_array(const unsigned int num_blocks)
{
    int *d_data1;
    const int syncArrLength = num_blocks;     //not using get_barrier_synch_array_length(num_blocks) as we dont need threadFenceReduction any more
    printf( "length of barrier synchronization array: %d\n", syncArrLength);
    unsigned int memReq=get_gmem_length<int>(num_blocks);
    printf( "device memory required for storing %d synchronizer vars is: %d\n", num_blocks,memReq);
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data1, memReq));
    int *d_syncArr=d_data1;
    printf("d_data1:%u, d_syncArr: %u \n",d_data1,d_syncArr);

    hipDeviceSynchronize();                                                                            //hipMalloc is asynchronous. So it may return back to host even before the above allocations were over.

// << grid size i.e. no. of blocks, block size i.e. no. of threads,  no. of bytes in shared mem. that is dynamically allocated per block for this call in addition to the statically allocated mem. >>
    initialize <<< dim3(1 + ((syncArrLength - 1) >> LOG_BLOCKDIM) ), dim3(BLOCKDIM), 0 >>> (syncArrLength,d_syncArr, -1);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel execution failed");                                                         // check if kernel execution generated an error
    return d_syncArr;
}


////////////////////////////////////////////////////////////////////
//! Allocate space for storing the sum and count for each cluster for every block.
//! For each block only first k threads store the reduced result in device memory.
//! Also each block reuses the device memory space of the block it reduced with.
//!
//! The sum and count values are initialised as 0 for all clusters of all blocks.
//! @return Base address of the count array which was created in the device memory.
///////////////////////////////////////////////////////////////////
__host__ int* allocate_initialise_reduction_result(const unsigned int num_blocks, const unsigned int k, float * &d_sumx, float * &d_sumy )
{
// For each block only first k threads store the reduced result in device memory.
// Also each block reuses the device memory space of the block it reduced with.
    int reduction_array_len = k*((num_blocks+1)>>1);

    float *d_data;
    unsigned int memReq=get_gmem_length<float>(reduction_array_len);
    printf( "device memory required for storing reduction result for centroid co-ordinates of k*(block_num/2) clusters (%d) is: %d\n", reduction_array_len,(memReq<<1));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data, (memReq<<1)));

    unsigned int lenPerDim_d_sum=memReq/sizeof(float);
    d_sumx=d_data;                                                                           //Stores the per coordinate sum of all the points in each cluster for each block
    d_sumy=d_sumx+lenPerDim_d_sum;
    printf("d_data:%u, d_sumx: %u, d_sumy:%u \n",d_data,d_sumx,d_sumy);


    int* d_data1;
    memReq=get_gmem_length<int>(reduction_array_len);
    printf( "device memory required for storing reduction result of count of points in k*(block_num/2) clusters (%d ) is: %d\n", reduction_array_len,memReq);
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data1, memReq));
    int *const d_count=d_data1;                                                                           //Stores the no. of points in each cluster for each block
    printf("d_data1:%u, d_count: %u \n",d_data1,d_count);

    // Initialize the sum variables to 0. Actually we should have initialised only reduction_array_len number of values for each attribute,
    // but since that would have meant calling initialise kernel 'number  of attribue' times, we set to 0 even the extra address
    // that we had allocated for each dimension to align with transactino size.
    initialize <<< dim3(1 + ((reduction_array_len - 1) >> LOG_BLOCKDIM)), dim3(BLOCKDIM), 0 >>> (reduction_array_len,d_sumx, 0);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel execution failed");                      // check if kernel execution generated an error
    initialize <<< dim3(1 + ((reduction_array_len - 1) >> LOG_BLOCKDIM)), dim3(BLOCKDIM), 0 >>> (reduction_array_len,d_sumy, 0);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel execution failed");                      // check if kernel execution generated an error
    initialize <<< dim3(1 + ((reduction_array_len - 1) >> LOG_BLOCKDIM)), dim3(BLOCKDIM), 0 >>> (reduction_array_len,d_count, 0);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel execution failed");                      // check if kernel execution generated an error

    return d_count;
}


////////////////////////////////////////////////////////////////////
//! Allocate space for storing the final cluster number every point belongs to.
//! It also initialises cluster no. for every point as 0 i.e. first cluster
//! @return Base address of the array which was created in the device memory.
///////////////////////////////////////////////////////////////////
__host__ int* allocate_initialise_clusterno_array(const unsigned int n)
{
    int * d_data1;
    unsigned int memReq=get_gmem_length<int>(n);
    printf( "device memory required for storing the cluster nos of %d data points is: %d\n", n,memReq);
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_data1, memReq ));
    int *const d_clusterno=d_data1;                                                                       //stores the cluster no. to which the data-point belongs
    printf("d_data1:%u, d_clusterno: %u \n",d_data1,d_clusterno);

    hipDeviceSynchronize();                                                                            //hipMalloc is asynchronous. So it may return back to host even before the above allocations were over.

// << grid size i.e. no. of blocks, block size i.e. no. of threads,  no. of bytes in shared mem. that is dynamically allocated per block for this call in addition to the statically allocated mem. >>
    initialize <<< dim3(1 + ((n - 1) >> LOG_BLOCKDIM) ), dim3(BLOCKDIM), 0 >>> (n,d_clusterno, 0);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel execution failed");                                                         // check if kernel execution generated an error
    return d_clusterno;
}


////////////////////////////////////////////////////////////////////////////////
//! Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    int n,max_iter,k;
    float threshold;

    float *h_dataptx=NULL;
    float *h_datapty=NULL;

    try
    {
        readInput( n, max_iter, k, threshold, h_dataptx, h_datapty); //reads from a file nmaed by macro INPUTFILE in the pwd

        CUT_DEVICE_INIT(argc, argv); //Checks if system has cuda devices and reads cmdline arg for cuda device to be used (by default 0). If the device is CUDA enabled then prints the device name else prints error and exits code.

        mean_by_CUDA(n, max_iter, k, threshold, h_dataptx, h_datapty);
    }
    catch (...)
    {
        if (h_dataptx!=NULL)
            free(h_dataptx);
        if (h_datapty!=NULL)
            free(h_datapty);

        h_dataptx=NULL;
        h_datapty=NULL;
    }

        if (h_dataptx!=NULL)
            free(h_dataptx);
        if (h_datapty!=NULL)
            free(h_datapty);

        h_dataptx=NULL;
        h_datapty=NULL;

   // CUT_EXIT(argc, argv); //Prompt for press enter to exit and then flush error and out stream. To avoid it pass "--noprompt" on cmdline.
}

#if APPROACH == 1
////////////////////////////////////////////////////////////////////////////////
//!Compare kmeans on CPU and GPU
//! @param n number of points
//! @param h_max_iter number of iterations
//! @param k number of clusters
////////////////////////////////////////////////////////////////////////////////
__host__ void mean_by_CUDA( int n, int h_max_iter, int k, float threshold, float *h_dataptx, float *h_datapty)
{
    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    printf("Shared mem. req. on each multiprocessor is: \n MIN_NUM_BLOCKS_PER_MULTIPROC * ( BLOCKDIM *POINTS_PER_THREAD*12  +  k*8)\n");
    printf("%d * ( %d*%d*12 + %d*8 ) = %d bytes\n", MIN_NUM_BLOCKS_PER_MULTIPROC, BLOCKDIM, POINTS_PER_THREAD, k, MIN_NUM_BLOCKS_PER_MULTIPROC * ( BLOCKDIM * POINTS_PER_THREAD * 12  +  k*8));

    bool load_points_in_cycles=false;
    int temp=get_num_blocks(n);
//    temp=1;
    if ((MIN_NUM_BLOCKS_PER_MULTIPROC * NUM_MULTIPROC) < temp)                                             //We cannot guarantee that all blocks will be started together, so this may lead to deadlock.
    {
      printf("\n%d blocks are needed whereas we can only allow a maximum of %d blocks to ensure that all blocks are started together.",temp,MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC);
      printf("\n We will load points in cycles to do work in %d blocks only.",MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC);
      printf("\nPlease increase number of threads or points per thread if you want to avoid this.\n");
      load_points_in_cycles=true;
      temp=MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC;
    }
    const int num_blocks=temp;

    printf( "no. of blocks: %d\n", num_blocks);
    //TODO: Right now there is no check if no. of blocks is more than the limit of 64k.
    //In that case we should be handling more than one point per thread by invoking the local reduction step multiple times.

    float *d_dataptx=NULL;                                                                          // x-coord of datapoints
    float *d_datapty=NULL;                                                                          // y-coord of datapoints
    unsigned int lenPerDim_d_datapt = load_points_in_device(n, d_dataptx, d_datapty,h_dataptx, h_datapty);

    //hipMemcpy is asynchronous. Required to make sure above memcopy is complete before we copy some of those values as centroids.
    hipDeviceSynchronize();

    float *d_centroidx=NULL;                                                                        // x-coord of centroids
    float *d_centroidy=NULL;                                                                        // y-coord of centroids
    unsigned int lenPerDim_d_centroid = load_centroids_in_device(k, d_centroidx, d_centroidy, d_dataptx, d_datapty);


    int *const d_syncArr=create_block_synchronisn_array(num_blocks);


    //Allocate device memory for result
    float *d_sumx=NULL;                                                                           //Stores the sum of x-coordinates of all the points in each cluster for each block
    float *d_sumy=NULL;                                                                           //Stores the sum of y-coordinates of all the points in each cluster for each block
    int *const d_count=allocate_initialise_reduction_result(num_blocks, k, d_sumx, d_sumy);      //Stores the no. of points in each cluster for each block


    int *const d_clusterno=allocate_initialise_clusterno_array(n);                               //stores the cluster no. to which the data-point belongs


    int *d_max_iter;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_max_iter, sizeof(int) ));
    hipDeviceSynchronize();                                                                     //hipMalloc is asynchronous. So it may return back to host even before the above allocations were over.
    CUDA_SAFE_CALL( hipMemcpy( d_max_iter, &h_max_iter, sizeof(int), hipMemcpyHostToDevice) );


    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "GPU version mem. allocation and copy time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutResetTimer( timer));


#if PROFILE_TIME == 1
    hipEvent_t start_cluster,stop_cluster;
    float time,time_cluster;
    time=time_cluster=0;

    hipEventCreate(&start_cluster);
    hipEventCreate(&stop_cluster);
#endif


    CUT_SAFE_CALL( cutStartTimer( timer));

#if PROFILE_TIME == 1
    hipEventRecord( start_cluster, 0 );
#endif
// << grid size i.e. no. of blocks, block size i.e. no. of threads,  no. of bytes in shared mem. that is dynamically allocated per block for this call in addition to the statically allocated mem. >>
    if(load_points_in_cycles)//load_points_in_cycles)
      //cluster1_load_optimized <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
      cluster1 <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
    else
      cluster <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
//  testBRDCST <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)) >>> (n, k, d_dataptx);

#if PROFILE_TIME == 1
    hipEventRecord( stop_cluster, 0 );
    hipEventSynchronize(stop_cluster);
    hipEventElapsedTime( &time, start_cluster, stop_cluster );
    time_cluster+=time;
    time=0;
#else
    hipDeviceSynchronize();
#endif
    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    if(load_points_in_cycles)
    {
      //findCluster <<<  dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_dataptx, d_datapty, d_centroidx, d_centroidy, d_clusterno);
      hipDeviceSynchronize();
      // check if kernel execution generated an error
      CUT_CHECK_ERROR("Kernel execution failed");
    }
    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "GPU version processing time: %f (ms)\n", cutGetTimerValue( timer));

#if PROFILE_TIME == 1
    printf("Total time in kernel: cluster = %f(ms)\n",time_cluster);

    hipEventDestroy(start_cluster);
    hipEventDestroy(stop_cluster);
#endif
    CUT_SAFE_CALL( cutResetTimer( timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    float *h_centroidx,*h_centroidy;
    h_centroidx=NULL;
    h_centroidy=NULL;
    int *h_clusterno=NULL;
    try
    {
        h_centroidx = (float*) malloc(sizeof(float)*k);
        h_centroidy = (float*) malloc(sizeof(float)*k);

        CUDA_SAFE_CALL( hipMemcpy( h_centroidx, d_centroidx, sizeof(float)*k, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( h_centroidy, d_centroidy, sizeof(float)*k, hipMemcpyDeviceToHost) );

        h_clusterno = (int*) malloc(sizeof(int)*n);

        CUDA_SAFE_CALL( hipMemcpy( h_clusterno, d_clusterno, sizeof(int)*n, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( &h_max_iter, d_max_iter, sizeof(int), hipMemcpyDeviceToHost) );


        CUDA_SAFE_CALL(hipFree(d_dataptx));
        CUDA_SAFE_CALL(hipFree(d_centroidx));
        CUDA_SAFE_CALL(hipFree(d_clusterno));
        CUDA_SAFE_CALL(hipFree(d_sumx) );
        CUDA_SAFE_CALL(hipFree(d_count) );
        CUDA_SAFE_CALL(hipFree(d_syncArr) );

        CUT_SAFE_CALL( cutStopTimer( timer));
        printf( "GPU version copying results back and deallocation time: %f (ms)\n", cutGetTimerValue( timer));
        CUT_SAFE_CALL( cutDeleteTimer( timer));

        printf( "GPU version number of iterations: %d\n", h_max_iter);

        writeOutput(n, k, h_dataptx, h_datapty, h_clusterno, h_centroidx, h_centroidy);
    }
    catch(...)
    {
        if(h_centroidx!=NULL)
            free(h_centroidx);
        if(h_centroidy!=NULL)
            free(h_centroidy);
        if(h_clusterno!=NULL)
            free(h_clusterno);

        h_centroidx=NULL;
        h_centroidy=NULL;
        h_clusterno=NULL;
    }

    if(h_centroidx!=NULL)
        free(h_centroidx);
    if(h_centroidy!=NULL)
        free(h_centroidy);
    if(h_clusterno!=NULL)
        free(h_clusterno);

    h_centroidx=NULL;
    h_centroidy=NULL;
    h_clusterno=NULL;
}
#endif

#if APPROACH == 155
////////////////////////////////////////////////////////////////////////////////
//!Compare kmeans on CPU and GPU
//! @param n number of points
//! @param h_max_iter number of iterations
//! @param k number of clusters
////////////////////////////////////////////////////////////////////////////////
__host__ void mean_by_CUDA( int n, int h_max_iter, int k, float threshold, float *h_dataptx, float *h_datapty)
{
    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    printf("Shared mem. req. on each multiprocessor is: \n MIN_NUM_BLOCKS_PER_MULTIPROC * ( BLOCKDIM *POINTS_PER_THREAD*12  +  k*8)\n");
    printf("%d * ( %d*%d*12 + %d*8 ) = %d bytes\n", MIN_NUM_BLOCKS_PER_MULTIPROC, BLOCKDIM, POINTS_PER_THREAD, k, MIN_NUM_BLOCKS_PER_MULTIPROC * ( BLOCKDIM * POINTS_PER_THREAD * 12  +  k*8));

    bool load_points_in_cycles=false;
    int temp=get_num_blocks(n);
//    temp=1;
    if ((MIN_NUM_BLOCKS_PER_MULTIPROC * NUM_MULTIPROC) < temp)                                             //We cannot guarantee that all blocks will be started together, so this may lead to deadlock.
    {
      printf("\n%d blocks are needed whereas we can only allow a maximum of %d blocks to ensure that all blocks are started together.",temp,MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC);
      printf("\n We will load points in cycles to do work in %d blocks only.",MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC);
      printf("\nPlease increase number of threads or points per thread if you want to avoid this.\n");
      load_points_in_cycles=true;
      temp=MIN_NUM_BLOCKS_PER_MULTIPROC*NUM_MULTIPROC;
    }
    const int num_blocks=temp;

    printf( "no. of blocks: %d\n", num_blocks);
    //TODO: Right now there is no check if no. of blocks is more than the limit of 64k.
    //In that case we should be handling more than one point per thread by invoking the local reduction step multiple times.

    float *d_dataptx=NULL;                                                                          // x-coord of datapoints
    float *d_datapty=NULL;                                                                          // y-coord of datapoints
    unsigned int lenPerDim_d_datapt = load_points_in_device(n, d_dataptx, d_datapty,h_dataptx, h_datapty);

    //hipMemcpy is asynchronous. Required to make sure above memcopy is complete before we copy some of those values as centroids.
    hipDeviceSynchronize();

    float *d_centroidx=NULL;                                                                        // x-coord of centroids
    float *d_centroidy=NULL;                                                                        // y-coord of centroids
    unsigned int lenPerDim_d_centroid = load_centroids_in_device(k, d_centroidx, d_centroidy, d_dataptx, d_datapty);


    int *const d_syncArr=create_block_synchronisn_array(num_blocks);


    //Allocate device memory for result
    float *d_sumx=NULL;                                                                           //Stores the sum of x-coordinates of all the points in each cluster for each block
    float *d_sumy=NULL;                                                                           //Stores the sum of y-coordinates of all the points in each cluster for each block
    int *const d_count=allocate_initialise_reduction_result(num_blocks, k, d_sumx, d_sumy);      //Stores the no. of points in each cluster for each block


    int *const d_clusterno=allocate_initialise_clusterno_array(n);                               //stores the cluster no. to which the data-point belongs


    int *d_max_iter;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_max_iter, sizeof(int) ));
    hipDeviceSynchronize();                                                                     //hipMalloc is asynchronous. So it may return back to host even before the above allocations were over.
    CUDA_SAFE_CALL( hipMemcpy( d_max_iter, &h_max_iter, sizeof(int), hipMemcpyHostToDevice) );


    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "GPU version mem. allocation and copy time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutResetTimer( timer));


#if PROFILE_TIME == 1
    hipEvent_t start_cluster,stop_cluster;
    float time,time_cluster;
    time=time_cluster=0;

    hipEventCreate(&start_cluster);
    hipEventCreate(&stop_cluster);
#endif


    CUT_SAFE_CALL( cutStartTimer( timer));

#if PROFILE_TIME == 1
    hipEventRecord( start_cluster, 0 );
#endif
// << grid size i.e. no. of blocks, block size i.e. no. of threads,  no. of bytes in shared mem. that is dynamically allocated per block for this call in addition to the statically allocated mem. >>
    if(load_points_in_cycles)//load_points_in_cycles)
    {
      cluster1_load_optimized_tree <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
      //cluster1_tree <<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
    }else{
      cluster_tree<<< dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_max_iter, d_dataptx,d_datapty,d_centroidx,d_centroidy,d_syncArr,d_sumx, d_sumy, d_count,d_clusterno);//, d_reducedCounts, d_timeVar);//,d_dbgIter);
    }

#if PROFILE_TIME == 1
    hipEventRecord( stop_cluster, 0 );
    hipEventSynchronize(stop_cluster);
    hipEventElapsedTime( &time, start_cluster, stop_cluster );
    time_cluster+=time;
    time=0;
#else
    hipDeviceSynchronize();
#endif
    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    if(load_points_in_cycles)
    {
      //findCluster <<<  dim3(num_blocks), dim3(BLOCKDIM), k*(sizeof(float)<<1) >>> (n, k, d_dataptx, d_datapty, d_centroidx, d_centroidy, d_clusterno);
      hipDeviceSynchronize();
      // check if kernel execution generated an error
      CUT_CHECK_ERROR("Kernel execution failed");
    }
    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "GPU version processing time: %f (ms)\n", cutGetTimerValue( timer));

#if PROFILE_TIME == 1
    printf("Total time in kernel: cluster = %f(ms)\n",time_cluster);

    hipEventDestroy(start_cluster);
    hipEventDestroy(stop_cluster);
#endif
    CUT_SAFE_CALL( cutResetTimer( timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    float *h_centroidx,*h_centroidy;
    h_centroidx=NULL;
    h_centroidy=NULL;
    int *h_clusterno=NULL;
    float *h_sumx,*h_sumy;
    h_sumx=NULL;
    h_sumy=NULL;
    int *h_count;
    h_count=NULL;
    try
    {
        h_centroidx = (float*) malloc(sizeof(float)*k);
        h_centroidy = (float*) malloc(sizeof(float)*k);

        CUDA_SAFE_CALL( hipMemcpy( h_centroidx, d_centroidx, sizeof(float)*k, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( h_centroidy, d_centroidy, sizeof(float)*k, hipMemcpyDeviceToHost) );

        h_clusterno = (int*) malloc(sizeof(int)*n);

        CUDA_SAFE_CALL( hipMemcpy( h_clusterno, d_clusterno, sizeof(int)*n, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( &h_max_iter, d_max_iter, sizeof(int), hipMemcpyDeviceToHost) );

        h_sumx = (float *)malloc(sizeof(float)*(k*((num_blocks+1)>>1)));

        h_sumy = (float *)malloc(sizeof(float)*(k*((num_blocks+1)>>1)));

        h_count = (int *)malloc(sizeof(int)*(k*((num_blocks+1)>>1)));

        CUDA_SAFE_CALL( hipMemcpy( h_sumx, d_sumx, sizeof(float)*(k*((num_blocks+1)>>1)), hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( h_sumy, d_sumy, sizeof(float)*(k*((num_blocks+1)>>1)), hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( h_count, d_count, sizeof(int)*(k*((num_blocks+1)>>1)), hipMemcpyDeviceToHost) );


        CUDA_SAFE_CALL(hipFree(d_dataptx));
        CUDA_SAFE_CALL(hipFree(d_centroidx));
        CUDA_SAFE_CALL(hipFree(d_clusterno));
        CUDA_SAFE_CALL(hipFree(d_sumx) );
        CUDA_SAFE_CALL(hipFree(d_count) );
        CUDA_SAFE_CALL(hipFree(d_syncArr) );

        CUT_SAFE_CALL( cutStopTimer( timer));
        printf( "GPU version copying results back and deallocation time: %f (ms)\n", cutGetTimerValue( timer));
        CUT_SAFE_CALL( cutDeleteTimer( timer));

        printf( "GPU version number of iterations: %d\n", h_max_iter);

        writeOutput(n, k, h_dataptx, h_datapty, h_clusterno, h_centroidx, h_centroidy);

        printf("\n %d sums and count are:\n",k);
        int sum_count=0;
        for ( int i = 0; i < k; ++i){
              printf("%f %f %d\n", *(h_sumx+i), *(h_sumy+i), *(h_count+i));
              sum_count+=(*(h_count+i));
        }
        if(sum_count!=n)
          printf("\nPanga total count did not match:%d",sum_count);
    }
    catch(...)
    {
        if(h_centroidx!=NULL)
            free(h_centroidx);
        if(h_centroidy!=NULL)
            free(h_centroidy);
        if(h_clusterno!=NULL)
            free(h_clusterno);

        if(h_sumx!=NULL)
            free(h_sumx);
        if(h_sumy!=NULL)
            free(h_sumy);
        if(h_count!=NULL)
            free(h_count);

        h_sumx=NULL;
        h_sumy=NULL;
        h_count=NULL;

        h_centroidx=NULL;
        h_centroidy=NULL;
        h_clusterno=NULL;
    }

    if(h_centroidx!=NULL)
        free(h_centroidx);
    if(h_centroidy!=NULL)
        free(h_centroidy);
    if(h_clusterno!=NULL)
        free(h_clusterno);

    if(h_sumx!=NULL)
        free(h_sumx);
    if(h_sumy!=NULL)
        free(h_sumy);
    if(h_count!=NULL)
        free(h_count);

    h_sumx=NULL;
    h_sumy=NULL;
    h_count=NULL;

    h_centroidx=NULL;
    h_centroidy=NULL;
    h_clusterno=NULL;
}
#endif
